#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define size 1073741824
#define threadsize 1024
#define partitions 2

__global__
void MatrixAddition(int *a, int *b, int *c){
    //1d grid with 2d blocks
    int id = blockIdx.x *blockDim.x * blockDim.y + threadIdx.y * blockDim.x+ threadIdx.x;

    c[id] = a[id] + b[id];
}

int main(){

    const long long int totalsize = size*sizeof(int);
    
    long long int summation = 0;

    float time1, time2 = 0.0;

    int *matA = (int*)malloc(totalsize);
    int *matB = (int*)malloc(totalsize);
    int *matC = (int*)malloc (totalsize);

    for(int i = 0; i < size;i++){
        matA[i] = 1;
        matB[i] = 2;
       // matC[i] = 0;
    }


    int *matAD; int *matBD; int *matCD;
    hipMalloc((void**)&matAD,totalsize/partitions);
    hipMalloc((void**)&matBD, totalsize/partitions);
    hipMalloc((void**)&matCD, totalsize/partitions);

    hipEvent_t start1,end1;
    hipEventCreate(&start1);
    hipEventCreate(&end1);

    //computing by partitions
   for(int k= 0; k<partitions; k++){

    hipMemcpy(matAD, &matA[k*size/partitions], totalsize/partitions, hipMemcpyHostToDevice);
    hipMemcpy(matBD, &matB[k*size/partitions], totalsize/partitions, hipMemcpyHostToDevice);

    dim3 dimGrid(size/threadsize/partitions,1);
    dim3 dimBlock(32,32);
    hipEventRecord(start1);
    MatrixAddition<<<dimGrid, dimBlock>>>(matAD,matBD, matCD);
    hipEventRecord(end1);
    hipMemcpy(&matC[k*size/partitions], matCD, totalsize/partitions, hipMemcpyDeviceToHost);
    hipEventSynchronize(end1);
    hipEventElapsedTime(&time1,start1,end1);

    time2+= time1;

}

printf("elapsed time is %lf milli secs \n",time2);
for(int i = 0; i < size; i++){
    summation += matC[i];
}
    printf("Sum is %lld ", summation);
    hipFree(matAD);
    hipFree(matBD);
    hipFree(matCD);
}