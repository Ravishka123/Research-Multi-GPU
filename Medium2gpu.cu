#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
//1073741824
#define size 16384
#define threadsize 1024

__global__
void MatrixAddition(int *a, int *b, int *c){
    int id = blockIdx.x *blockDim.x * blockDim.y + threadIdx.y * blockDim.x+ threadIdx.x;


    c[id] = a[id] + b[id];
}

int main(){

    const long long int totalsize = size*sizeof(int);
    
    long long int summation = 0;
    float time1, time2 = 0.0;

    int *matA = (int*)malloc(totalsize);
    int *matB = (int*)malloc(totalsize);
    int *matC = (int*)malloc (totalsize);

    for(int i = 0; i < size;i++){
        matA[i] = 1;
        matB[i] = 2;
       matC[i] = 0;
    }

    
    
    
    dim3 dimGrid(size/threadsize/2,1);
    dim3 dimBlock(32,32);

    hipStream_t stream[2];
    
    int *matAD[2]; int *matBD[2]; int *matCD[2];
    int *matAP;int *matBP; int *matCP;
    hipHostMalloc((void**)&matAP,totalsize, hipHostMallocDefault);
    hipHostMalloc((void**)&matBP,totalsize, hipHostMallocDefault);
    hipHostMalloc((void**)&matCP,totalsize, hipHostMallocDefault);
    memcpy(matAP,matA, totalsize);
    memcpy(matBP,matB, totalsize);
   memcpy(matCP,matC, totalsize);

    
    hipSetDevice(0);
    
    hipMalloc((void**)&matAD[0],totalsize/2);
    hipMalloc((void**)&matBD[0], totalsize/2);
    hipMalloc((void**)&matCD[0], totalsize/2);
    hipStreamCreateWithFlags(&stream[0],hipStreamNonBlocking);
   

    hipSetDevice(1);
  
    hipMalloc((void**)&matAD[1],totalsize/2);
    hipMalloc((void**)&matBD[1], totalsize/2);
    hipMalloc((void**)&matCD[1], totalsize/2);
  
    
    hipStreamCreateWithFlags(&stream[1],hipStreamNonBlocking);
   
    
   
   
    hipSetDevice(0);
    hipMemcpyAsync(matAD[0], &matAP[0*size/2], totalsize/2, hipMemcpyHostToDevice,stream[0]);
    hipMemcpyAsync(matBD[0], &matBP[0*size/2], totalsize/2, hipMemcpyHostToDevice,stream[0]); 
    hipSetDevice(1);
    hipMemcpyAsync(matAD[1], &matAP[1*size/2], totalsize/2, hipMemcpyHostToDevice,stream[1]);
    hipMemcpyAsync(matBD[1], &matBP[1*size/2], totalsize/2, hipMemcpyHostToDevice,stream[1]);

  
    hipSetDevice(0);
    MatrixAddition<<<dimGrid, dimBlock,0,stream[0]>>>(matAD[0],matBD[0], matCD[0]);
   hipSetDevice(1);
    MatrixAddition<<<dimGrid, dimBlock,0,stream[1]>>>(matAD[1],matBD[1], matCD[1]);
   

    hipSetDevice(0);
    hipMemcpyAsync(&matCP[0*size/2], matCD[0], totalsize/2, hipMemcpyDeviceToHost,stream[0]);
    hipSetDevice(1);
    hipMemcpyAsync(&matCP[1*size/2], matCD[1], totalsize/2, hipMemcpyDeviceToHost,stream[1]);
   
    memcpy(matC, matCP, totalsize);
   

   

for(int i = 0; i < size; i++){
    summation += matCP[i];
}
    printf("Sum is %lld ", summation);
    hipFree(matAD);
    hipFree(matBD);
    hipFree(matCD);
    hipHostFree(matAP);
    hipHostFree(matBP);
    hipHostFree(matCP);
}