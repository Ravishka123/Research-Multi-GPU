#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define size 16384
#define threadsize 1024


__global__
void MatrixAddition(int *a, int *b, int *c){
    //1d grid with 2d blocks
    int id = blockIdx.x *blockDim.x * blockDim.y + threadIdx.y * blockDim.x+ threadIdx.x;
    c[id] = a[id] + b[id];
}

int main(){

    const long long int totalsize = size*sizeof(int);
    
    long long int summation = 0;

    float time1 = 0.0;

    int *matA = (int*)malloc(totalsize);
    int *matB = (int*)malloc(totalsize);
    int *matC = (int*)malloc (totalsize);

    for(int i = 0; i < size;i++){
        matA[i] = 1;
        matB[i] = 2;
       // matC[i] = 0;
    }


    int *matAD; int *matBD; int *matCD;
    hipMalloc((void**)&matAD,totalsize);
    hipMalloc((void**)&matBD, totalsize);
    hipMalloc((void**)&matCD, totalsize);

    hipEvent_t start1,end1;
    hipEventCreate(&start1);
    hipEventCreate(&end1);

    
   

    hipMemcpy(matAD, matA, totalsize, hipMemcpyHostToDevice);
    hipMemcpy(matBD, matB, totalsize, hipMemcpyHostToDevice);

    dim3 dimGrid(size/threadsize,1);
    dim3 dimBlock(32,32);
    hipEventRecord(start1);
    MatrixAddition<<<dimGrid, dimBlock>>>(matAD,matBD, matCD);
    hipEventRecord(end1);
    hipMemcpy(matC, matCD, totalsize, hipMemcpyDeviceToHost);
    hipEventSynchronize(end1);
    hipEventElapsedTime(&time1,start1,end1);

    



printf("elapsed time is %lf milli secs \n",time1);
for(int i = 0; i < size; i++){
    summation += matC[i];
}
    printf("Sum is %lld ", summation);
    hipFree(matAD);
    hipFree(matBD);
    hipFree(matCD);
}